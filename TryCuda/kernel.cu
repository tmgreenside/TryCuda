﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <memory>
#include <random>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void init_random(const int size, double* array_in);

bool mult_matrices(double* c, const double* a, const double* b, 
    const int size_a_x, const int size_a_y, 
    const int size_b_x, const int size_b_y, 
    const int size_c)
{
    bool success = true;

    if ((size_c < (size_a_x * size_b_y)) || (size_a_x != size_b_y))
    {
        success = false;
    }

    if (success)
    {

    }

    return success;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    // Create two matrices, multiply them, get result. Get the time difference between CPU and with CUDA.
    int size_x = 500;
    int size_y = 500;
    shared_ptr<double> matrix_A(new double[size_x * size_y]);
    shared_ptr<double> matrix_B(new double[size_x * size_y]);

    init_random(size_x * size_y, matrix_A.get());
    init_random(size_x * size_y, matrix_B.get());


}

void init_random(const int size, double* array_in)
{
    for (int i = 0; i < size; i++)
    {
        array_in[i] = (double)rand();
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
